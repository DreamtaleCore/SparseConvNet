
#include <hip/hip_runtime.h>
// Copyright 2016-present, Facebook, Inc.
// All rights reserved.
//
// This source code is licensed under the license found in the
// LICENSE file in the root directory of this source tree.

#ifndef TH_GENERIC_FILE_
#define TH_GENERIC_FILE_ "generic/GPU/MaxPooling.cu"
#else
#include "MaxPooling.h"
#include "RuleBookIterator.h"

extern "C" void scn_DR_(MaxPooling_updateOutput)(
    THLongTensor *inputSize, THLongTensor *outputSize, THLongTensor *poolSize,
    THLongTensor *poolStride, void **m, THCTensor *input_features,
    THCTensor *output_features, long nFeaturesToDrop) {

  SCN_INITIALIZE_AND_REFERENCE(Metadata<Dimension>, m)
  uInt nPlanes = input_features->size[1] - nFeaturesToDrop;
  auto _rules =
      _m.getRuleBook(inputSize, outputSize, poolSize, poolStride, true);
  uInt nActive = _m.getNActive(outputSize);
  THCTensor_(resize2d)(state, output_features, nActive, nPlanes);
  THCTensor_(zero)(state, output_features);

  auto iF = THCTensor_(data)(state, input_features) + nFeaturesToDrop;
  auto oF = THCTensor_(data)(state, output_features);
  RULEBOOKITERATOR(
      MaxPooling_ForwardPass<real>(THCState_getCurrentStream(state), iF, oF,
                                   nPlanes, input_features->size[1],
                                   output_features->size[1], rbB, nHotB);
      , )
}
extern "C" void scn_DR_(MaxPooling_updateGradInput)(
    THLongTensor *inputSize, THLongTensor *outputSize, THLongTensor *poolSize,
    THLongTensor *poolStride, void **m, THCTensor *input_features,
    THCTensor *d_input_features, THCTensor *output_features,
    THCTensor *d_output_features, long nFeaturesToDrop) {

  SCN_INITIALIZE_AND_REFERENCE(Metadata<Dimension>, m)
  uInt nPlanes = input_features->size[1] - nFeaturesToDrop;
  auto _rules =
      _m.getRuleBook(inputSize, outputSize, poolSize, poolStride, true);
  uInt nActive = _m.getNActive(outputSize);
  THCTensor_(resizeAs)(state, d_input_features, input_features);
  THCTensor_(zero)(state, d_input_features);

  auto iF = THCTensor_(data)(state, input_features);
  auto oF = THCTensor_(data)(state, output_features);
  auto diF = THCTensor_(data)(state, d_input_features);
  auto doF = THCTensor_(data)(state, d_output_features);
  RULEBOOKITERATOR(
      MaxPooling_BackwardPass<real>(THCState_getCurrentStream(state), iF, diF,
                                    oF, doF, nPlanes, input_features->size[1],
                                    d_output_features->size[1], rbB, nHotB);
      , )
}
extern "C" void scn_DR_(RandomizedStrideMaxPooling_updateOutput)(
    THLongTensor *inputSize, THLongTensor *outputSize, THLongTensor *poolSize,
    THLongTensor *poolStride, void **m, THCTensor *input_features,
    THCTensor *output_features, long nFeaturesToDrop) {

  SCN_INITIALIZE_AND_REFERENCE(Metadata<Dimension>, m)
  uInt nPlanes = input_features->size[1] - nFeaturesToDrop;
  auto _rules =
      _m.getRandomizedStrideRuleBook(inputSize, outputSize, poolSize, poolStride, true);
  uInt nActive = _m.getNActive(outputSize);
  THCTensor_(resize2d)(state, output_features, nActive, nPlanes);
  THCTensor_(zero)(state, output_features);

  auto iF = THCTensor_(data)(state, input_features) + nFeaturesToDrop;
  auto oF = THCTensor_(data)(state, output_features);
  RULEBOOKITERATOR(
      MaxPooling_ForwardPass<real>(THCState_getCurrentStream(state), iF, oF,
                                   nPlanes, input_features->size[1],
                                   output_features->size[1], rbB, nHotB);
      , )
}
extern "C" void scn_DR_(RandomizedStrideMaxPooling_updateGradInput)(
    THLongTensor *inputSize, THLongTensor *outputSize, THLongTensor *poolSize,
    THLongTensor *poolStride, void **m, THCTensor *input_features,
    THCTensor *d_input_features, THCTensor *output_features,
    THCTensor *d_output_features, long nFeaturesToDrop) {

  SCN_INITIALIZE_AND_REFERENCE(Metadata<Dimension>, m)
  uInt nPlanes = input_features->size[1] - nFeaturesToDrop;
  auto _rules =
      _m.getRandomizedStrideRuleBook(inputSize, outputSize, poolSize, poolStride, true);
  uInt nActive = _m.getNActive(outputSize);
  THCTensor_(resizeAs)(state, d_input_features, input_features);
  THCTensor_(zero)(state, d_input_features);

  auto iF = THCTensor_(data)(state, input_features);
  auto oF = THCTensor_(data)(state, output_features);
  auto diF = THCTensor_(data)(state, d_input_features);
  auto doF = THCTensor_(data)(state, d_output_features);
  RULEBOOKITERATOR(
      MaxPooling_BackwardPass<real>(THCState_getCurrentStream(state), iF, diF,
                                    oF, doF, nPlanes, input_features->size[1],
                                    d_output_features->size[1], rbB, nHotB);
      , )
}
#endif
