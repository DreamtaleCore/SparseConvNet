
#include <hip/hip_runtime.h>
// Copyright 2016-present, Facebook, Inc.
// All rights reserved.
//
// This source code is licensed under the license found in the
// LICENSE file in the root directory of this source tree.

#ifndef TH_GENERIC_FILE_
#define TH_GENERIC_FILE_ "generic/GPU/Convolution.cu"
#else
#include "Convolution.h"
#include "RuleBookIterator.h"
#include <algorithm>
#include <cstring>

extern "C" double scn_DR_(Convolution_updateOutput)(
    THLongTensor *inputSize, THLongTensor *outputSize, THLongTensor *filterSize,
    THLongTensor *filterStride, void **m, THCTensor *input_features,
    THCTensor *output_features, THCTensor *weight, THCTensor *bias,
    long filterVolume) {
  SCN_INITIALIZE_AND_REFERENCE(Metadata<Dimension>, m)
  auto _rules =
      _m.getRuleBook(inputSize, outputSize, filterSize, filterStride, true);
  uInt nActive = _m.getNActive(outputSize);
  THCTensor_(resize2d)(state, output_features, nActive, weight->size[1]);
  if (not bias)
    THCTensor_(zero)(state, output_features);

  double flops = 0;
  if (nActive) {
    auto iF = THCTensor_(data)(state, input_features);
    auto oF = THCTensor_(data)(state, output_features);
    auto ip = input_features->size[1];
    auto op = output_features->size[1];
    auto w = THCTensor_(data)(state, weight);

    if (bias) {
      auto b = THCTensor_(data)(state, bias);
      for (uInt i = 0; i < op; i += 32) {
        uInt blockDim = min(32L, op - i);
        uInt gridDim = min(4096, nActive);
        Convolution_fp_bias<<<gridDim, blockDim, 0,
                              THCState_getCurrentStream(state)>>>(
            oF + i, b + i, op, op, nActive);
      }
    }
    uInt c = ip * op;
    RULEBOOKITERATOR(
        dConvolution_forward2<real>(iF, oF, w, rbB, nHotB, ip, ip, op, op,
                                    THCState_getCurrentStream(state));
        , w += c; flops += nHotB * c;)
  }
  return flops;
}

extern "C" void scn_DR_(Convolution_backward)(
    THLongTensor *inputSize, THLongTensor *outputSize, THLongTensor *filterSize,
    THLongTensor *filterStride, void **m, THCTensor *input_features,
    THCTensor *d_input_features, THCTensor *d_output_features,
    THCTensor *weight, THCTensor *d_weight, THCTensor *d_bias,
    long filterVolume) {
  SCN_INITIALIZE_AND_REFERENCE(Metadata<Dimension>, m)
  auto _rules =
      _m.getRuleBook(inputSize, outputSize, filterSize, filterStride, true);
  uInt nActive = _m.getNActive(outputSize);
  THCTensor_(resizeAs)(state, d_input_features, input_features);
  THCTensor_(zero)(state, d_input_features);

  if (nActive) {
    auto iF = THCTensor_(data)(state, input_features);
    auto diF = THCTensor_(data)(state, d_input_features);
    auto doF = THCTensor_(data)(state, d_output_features);
    auto ip = input_features->size[1];
    auto op = d_output_features->size[1];
    auto w = THCTensor_(data)(state, weight);
    auto dw = THCTensor_(data)(state, d_weight);
    uInt c = ip * op;
    RULEBOOKITERATOR(dConvolution_backward_dW2<real>(
                         iF, diF, doF, w, dw, rbB, nHotB, ip, ip, op, op,
                         THCState_getCurrentStream(state));
                     , w += c; dw += c;)

    if (d_bias) {
      auto db = THCTensor_(data)(state, d_bias);
      Convolution_bp_bias(doF, db, op, op, nActive,
                          THCState_getCurrentStream(state));
    }
  }
}

extern "C" double scn_DR_(SubmanifoldConvolution_updateOutput)(
    THLongTensor *inputSize, THLongTensor *filterSize, void **m,
    THCTensor *input_features, THCTensor *output_features, THCTensor *weight,
    THCTensor *bias, long filterVolume) {
  SCN_INITIALIZE_AND_REFERENCE(Metadata<Dimension>, m)
  auto _rules = _m.getSubmanifoldRuleBook(inputSize, filterSize, true);
  uInt nActive = _m.getNActive(inputSize);
  THCTensor_(resize2d)(state, output_features, nActive, weight->size[1]);
  if (not bias)
    THCTensor_(zero)(state, output_features);

  double flops = 0;
  if (nActive) {
    auto iF = THCTensor_(data)(state, input_features);
    auto oF = THCTensor_(data)(state, output_features);
    auto ip = input_features->size[1];
    auto op = output_features->size[1];
    auto w = THCTensor_(data)(state, weight);

    if (bias) {
      auto b = THCTensor_(data)(state, bias);
      for (uInt i = 0; i < op; i += 32) {
        uInt blockDim = min(32L, op - i);
        uInt gridDim = min(4096, nActive);
        Convolution_fp_bias<<<gridDim, blockDim, 0,
                              THCState_getCurrentStream(state)>>>(
            oF + i, b + i, op, op, nActive);
      }
    }
    uInt c = ip * op;
    RULEBOOKITERATOR(
        dConvolution_forward2<real>(iF, oF, w, rbB, nHotB, ip, ip, op, op,
                                    THCState_getCurrentStream(state));
        , w += c; flops += nHotB * c;)
  }
  return flops;
}

extern "C" void scn_DR_(SubmanifoldConvolution_backward)(
    THLongTensor *inputSize, THLongTensor *filterSize, void **m,
    THCTensor *input_features, THCTensor *d_input_features,
    THCTensor *d_output_features, THCTensor *weight, THCTensor *d_weight,
    THCTensor *d_bias, long filterVolume) {
  SCN_INITIALIZE_AND_REFERENCE(Metadata<Dimension>, m)
  auto _rules = _m.getSubmanifoldRuleBook(inputSize, filterSize, true);
  uInt nActive = _m.getNActive(inputSize);
  THCTensor_(resizeAs)(state, d_input_features, input_features);
  THCTensor_(zero)(state, d_input_features);

  if (nActive) {
    auto iF = THCTensor_(data)(state, input_features);
    auto diF = THCTensor_(data)(state, d_input_features);
    auto doF = THCTensor_(data)(state, d_output_features);
    auto ip = input_features->size[1];
    auto op = d_output_features->size[1];
    auto w = THCTensor_(data)(state, weight);
    auto dw = THCTensor_(data)(state, d_weight);
    uInt c = ip * op;
    RULEBOOKITERATOR(dConvolution_backward_dW2<real>(
                         iF, diF, doF, w, dw, rbB, nHotB, ip, ip, op, op,
                         THCState_getCurrentStream(state));
                     , w += c; dw += c;)

    if (d_bias) {
      auto db = THCTensor_(data)(state, d_bias);
      Convolution_bp_bias(doF, db, op, op, nActive,
                          THCState_getCurrentStream(state));
    }
  }
}

extern "C" double scn_DR_(FullConvolution_updateOutput)(
    THLongTensor *inputSize, THLongTensor *outputSize, THLongTensor *filterSize,
    THLongTensor *filterStride, void **mIn, void **mOut,
    THCTensor *input_features, THCTensor *output_features, THCTensor *weight,
    THCTensor *bias, long filterVolume, THCITensor *rulesBuffer) {

  SCN_INITIALIZE_AND_REFERENCE(Metadata<Dimension>, mIn)
  SCN_INITIALIZE_AND_REFERENCE(Metadata<Dimension>, mOut)
  auto _rules = _mIn.getFullConvolutionRuleBook(
      inputSize, outputSize, filterSize, filterStride, _mOut);
  uInt nActive = _mOut.getNActive(outputSize);
  THCTensor_(resize2d)(state, output_features, nActive, weight->size[1]);
  if (not bias)
    THCTensor_(zero)(state, output_features);
  double flops = 0;

  if (nActive) {
    auto iF = THCTensor_(data)(state, input_features);
    auto oF = THCTensor_(data)(state, output_features);
    auto ip = input_features->size[1];
    auto op = output_features->size[1];
    auto w = THCTensor_(data)(state, weight);

    if (bias) {
      auto b = THCTensor_(data)(state, bias);
      for (uInt i = 0; i < op; i += 32) {
        uInt blockDim = min(32L, op - i);
        uInt gridDim = min(4096, nActive);
        Convolution_fp_bias<<<gridDim, blockDim, 0,
                              THCState_getCurrentStream(state)>>>(
            oF + i, b + i, op, op, nActive);
      }
    }
    uInt c = ip * op;
    RULEBOOKITERATOR(
        dConvolution_forward2<real>(iF, oF, w, rbB, nHotB, ip, ip, op, op,
                                    THCState_getCurrentStream(state));
        , w += c; flops += nHotB * c;)
  }
  return flops;
}

extern "C" void scn_DR_(FullConvolution_backward)(
    THLongTensor *inputSize, THLongTensor *outputSize, THLongTensor *filterSize,
    THLongTensor *filterStride, void **mIn, void **mOut,
    THCTensor *input_features, THCTensor *d_input_features,
    THCTensor *d_output_features, THCTensor *weight, THCTensor *d_weight,
    THCTensor *d_bias, long filterVolume) {

  SCN_INITIALIZE_AND_REFERENCE(Metadata<Dimension>, mIn)
  SCN_INITIALIZE_AND_REFERENCE(Metadata<Dimension>, mOut)
  auto _rules = _mIn.getFullConvolutionRuleBook(
      inputSize, outputSize, filterSize, filterStride, _mOut);
  uInt nActive = _mOut.getNActive(outputSize);
  THCTensor_(resizeAs)(state, d_input_features, input_features);
  THCTensor_(zero)(state, d_input_features);
  if (nActive) {
    auto iF = THCTensor_(data)(state, input_features);
    auto diF = THCTensor_(data)(state, d_input_features);
    auto doF = THCTensor_(data)(state, d_output_features);
    auto ip = input_features->size[1];
    auto op = d_output_features->size[1];
    auto w = THCTensor_(data)(state, weight);
    auto dw = THCTensor_(data)(state, d_weight);
    uInt c = ip * op;
    RULEBOOKITERATOR(dConvolution_backward_dW2<real>(
                         iF, diF, doF, w, dw, rbB, nHotB, ip, ip, op, op,
                         THCState_getCurrentStream(state));
                     , w += c; dw += c;)

    if (d_bias) {
      auto db = THCTensor_(data)(state, d_bias);
      Convolution_bp_bias(doF, db, op, op, nActive,
                          THCState_getCurrentStream(state));
    }
  }
}
extern "C" double scn_DR_(RandomizedStrideConvolution_updateOutput)(
    THLongTensor *inputSize, THLongTensor *outputSize, THLongTensor *filterSize,
    THLongTensor *filterStride,
    void **m, THCTensor *input_features, THCTensor *output_features,
    THCTensor *weight, THCTensor *bias, long filterVolume,
    THCITensor *rulesBuffer) {
  SCN_INITIALIZE_AND_REFERENCE(Metadata<Dimension>, m)
  auto _rules =
      _m.getRandomizedStrideRuleBook(inputSize, outputSize, filterSize, filterStride, true);
  uInt nActive = _m.getNActive(outputSize);
  THCTensor_(resize2d)(state, output_features, nActive, weight->size[1]);
  if (not bias)
    THCTensor_(zero)(state, output_features);

  double flops = 0;
  if (nActive) {
    auto iF = THCTensor_(data)(state, input_features);
    auto oF = THCTensor_(data)(state, output_features);
    auto ip = input_features->size[1];
    auto op = output_features->size[1];
    auto w = THCTensor_(data)(state, weight);

    if (bias) {
      auto b = THCTensor_(data)(state, bias);
      for (uInt i = 0; i < op; i += 32) {
        uInt blockDim = min(32L, op - i);
        uInt gridDim = min(4096, nActive);
        Convolution_fp_bias<<<gridDim, blockDim, 0,
                              THCState_getCurrentStream(state)>>>(
            oF + i, b + i, op, op, nActive);
      }
    }
    uInt c = ip * op;
    RULEBOOKITERATOR(
        dConvolution_forward2<real>(iF, oF, w, rbB, nHotB, ip, ip, op, op,
                                    THCState_getCurrentStream(state));
        , w += c; flops += nHotB * c;)
  }
  return flops;
}

extern "C" void scn_DR_(RandomizedStrideConvolution_backward)(
    THLongTensor *inputSize, THLongTensor *outputSize, THLongTensor *filterSize,
    THLongTensor *filterStride,
    void **m, THCTensor *input_features, THCTensor *d_input_features,
    THCTensor *d_output_features, THCTensor *weight, THCTensor *d_weight,
    THCTensor *d_bias, long filterVolume) {
  SCN_INITIALIZE_AND_REFERENCE(Metadata<Dimension>, m)
  auto _rules =
      _m.getRandomizedStrideRuleBook(inputSize, outputSize, filterSize, filterStride, true);
  uInt nActive = _m.getNActive(outputSize);
  THCTensor_(resizeAs)(state, d_input_features, input_features);
  THCTensor_(zero)(state, d_input_features);

  if (nActive) {
    auto iF = THCTensor_(data)(state, input_features);
    auto diF = THCTensor_(data)(state, d_input_features);
    auto doF = THCTensor_(data)(state, d_output_features);
    auto ip = input_features->size[1];
    auto op = d_output_features->size[1];
    auto w = THCTensor_(data)(state, weight);
    auto dw = THCTensor_(data)(state, d_weight);
    uInt c = ip * op;
    RULEBOOKITERATOR(dConvolution_backward_dW2<real>(
                         iF, diF, doF, w, dw, rbB, nHotB, ip, ip, op, op,
                         THCState_getCurrentStream(state));
                     , w += c; dw += c;)

    if (d_bias) {
      auto db = THCTensor_(data)(state, d_bias);
      Convolution_bp_bias(doF, db, op, op, nActive,
                          THCState_getCurrentStream(state));
    }
  }
}
#endif
