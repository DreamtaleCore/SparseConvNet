
#include <hip/hip_runtime.h>
// Copyright 2016-present, Facebook, Inc.
// All rights reserved.
//
// This source code is licensed under the license found in the
// LICENSE file in the root directory of this source tree.

#ifndef TH_GENERIC_FILE_
#define TH_GENERIC_FILE_ "generic/GPU/SparseToDense.cu"
#else
#include "SparseToDense.h"

extern "C" void scn_DR_(SparseToDense_updateOutput)(
    THLongTensor *inputSize, void **m, THCTensor *input_features,
    THCTensor *output_features, long nPlanes) {

  SCN_INITIALIZE_AND_REFERENCE(Metadata<Dimension>, m)
  {
    long sz[Dimension + 2];
    sz[0] = _m.grids.begin()->second.size(); //batch size
    sz[1] = nPlanes;
    std::memcpy(sz + 2, THLongTensor_data(inputSize), sizeof(long) * Dimension);
    THCTensor_(resizeNd)(state, output_features, Dimension + 2, sz, NULL);
    THCTensor_(zero)(state, output_features);
  }
  if (input_features->nDimension == 2) {
    auto _rules = _m.getSparseToDenseRuleBook(inputSize, true);
    uInt _nPlanes = input_features->size[1];
    auto iF = THCTensor_(data)(state, input_features);
    auto oF = THCTensor_(data)(state, output_features);
    long spatialVolume = THLongTensor_prodall(inputSize);
    RULEBOOKITERATOR(
        SparseToDense_ForwardPass<real>(THCState_getCurrentStream(state), iF,
                                        oF, _nPlanes, spatialVolume, rbB, nHotB);
        , oF += _nPlanes * spatialVolume;)
  }
}
extern "C" void scn_DR_(SparseToDense_updateGradInput)(
    THLongTensor *inputSize, void **m, THCTensor *input_features,
    THCTensor *d_input_features, THCTensor *d_output_features) {

  SCN_INITIALIZE_AND_REFERENCE(Metadata<Dimension>, m)
  THCTensor_(resizeAs)(state, d_input_features, input_features);
  THCTensor_(zero)(state, d_input_features);

  if (input_features->nDimension == 2) {
    auto _rules = _m.getSparseToDenseRuleBook(inputSize, true);
    long spatialVolume = THLongTensor_prodall(inputSize);
    uInt _nPlanes = d_input_features->size[1];
    auto diF = THCTensor_(data)(state, d_input_features);
    auto doF = THCTensor_(data)(state, d_output_features);
    RULEBOOKITERATOR(SparseToDense_BackwardPass<real>(
                         THCState_getCurrentStream(state), diF, doF, _nPlanes,
                         spatialVolume, rbB, nHotB);
                     , doF += _nPlanes * spatialVolume;)
  }
}
#endif
